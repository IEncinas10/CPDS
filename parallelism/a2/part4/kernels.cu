#include <hip/hip_runtime.h>
#include <float.h>
#include <math.h>

__global__ void gpu_Heat(float *h, float *g, int N) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < 1 || x > N - 2 || y < 1 || y > N - 2)
	return;

    g[N * x + y] =
	0.25 * (h[N * x + y - 1] + h[N * x + y + 1] + h[N * (x - 1) + y] + h[N * (x + 1) + y]);
}

__global__ void gpu_Heat_diff(float *h, float *g, float *diff, int N) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < 1 || x > N - 2 || y < 1 || y > N - 2)
	return;

    g[N * x + y] =
	0.25 * (h[N * x + y - 1] + h[N * x + y + 1] + h[N * (x - 1) + y] + h[N * (x + 1) + y]);

    diff[(N - 2) * (x - 1) + y - 1] = (g[N * x + y] - h[N * x + y]) * (g[N * x + y] - h[N * x + y]);
}


#define MAX_THREADS_PER_BLOCK 2048

// idata: array of N elements.
// odata: output array of "GridDim" elements, 1 per block
__global__ void reduce(float *idata, float *odata, int N) {
    __shared__ float sdata[MAX_THREADS_PER_BLOCK];
    unsigned int s;

    // Cada thread realiza la suma parcial de los datos que le
    // corresponden y la deja en la memoria compartida
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    unsigned int gridSize = blockDim.x * 2 * gridDim.x;
    sdata[tid] = 0;
    while (i < N) {
	sdata[tid] += idata[i] + idata[i + blockDim.x];
	i += gridSize;
    }
    __syncthreads();

    // Hacemos la reduccion en la memoria compartida
    for (s = blockDim.x / 2; s > 32; s >>= 1) {
	if (tid < s)
	    sdata[tid] += sdata[tid + s];
	__syncthreads();
    }
    // desenrrollamos el ultimo warp activo
    if (tid < 32) {
	volatile float *smem = sdata;

	smem[tid] += smem[tid + 32];
	smem[tid] += smem[tid + 16];
	smem[tid] += smem[tid + 8];
	smem[tid] += smem[tid + 4];
	smem[tid] += smem[tid + 2];
	smem[tid] += smem[tid + 1];
    }

    // El thread 0 escribe el resultado de este bloque en la memoria global
    if (tid == 0)
	odata[blockIdx.x] = sdata[0];
}

__global__ void Kernel06(float *g_idata, float *g_odata) {
  __shared__ float sdata[MAX_THREADS_PER_BLOCK];
  unsigned int s;

  // Cada thread carga 2 elementos desde la memoria global,
  // los suma y los deja en la memoria compartida
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
  sdata[tid] = g_idata[i] + g_idata[i+blockDim.x];
  __syncthreads();

  // Hacemos la reduccion en la memoria compartida
  for (s=blockDim.x/2; s>32; s>>=1) {
    if (tid < s)
      sdata[tid] += sdata[tid + s];
    __syncthreads();
  }

 // desenrrollamos el ultimo warp activo
 if (tid < 32) {
   volatile float *smem = sdata;

   smem[tid] += smem[tid + 32];
   smem[tid] += smem[tid + 16];
   smem[tid] += smem[tid + 8];
   smem[tid] += smem[tid + 4];
   smem[tid] += smem[tid + 2];
   smem[tid] += smem[tid + 1];
 }

 // El thread 0 escribe el resultado de este bloque en la memoria global
 if (tid == 0) g_odata[blockIdx.x] = sdata[0];

}


