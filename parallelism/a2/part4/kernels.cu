#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>

__global__ void gpu_Heat (float *h, float *g, int N) {

	// TODO: kernel computation
	//...

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if(x < 1 || x > N - 2 || y < 1 || y > N - 2)
	    return;

	g[N * x + y] =  0.25 * (h[N * x       + y - 1] + 
				h[N * x       + y + 1] + 
				h[N * (x - 1) + y    ] + 
				h[N * (x + 1) + y    ]);
}
