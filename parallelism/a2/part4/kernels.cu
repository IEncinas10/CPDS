#include "hip/hip_runtime.h"
#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>

__global__ void gpu_Heat (float *h, float *g, int N) {

	// TODO: kernel computation
	//...

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if(x <= 0 || x >= N - 1 || y <= 0 || y >= N - 1)
	    return;

	g[N * x + y] =  0.25 * (f[N * x       + y - 1] + 
				f[N * x       + y + 1] + 
				f[N * (x - 1) + y    ] + 
				f[N * (x + 1) + y    ]);
}
